#include "hip/hip_runtime.h"
#include "cuda_kernels.hpp"

__global__ void restore_kernel(uchar* ret, const float* Yst, const float* mask,
        const uchar* Xt, const float* trans,
        const int H, const int W, const int h, const int w) {
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    int offset = blockDim.x * gridDim.x;
    while (tid < h * w) {
        int x = tid % w;
        int y = tid / w;
        float sx = x*trans[0] + y*trans[1] + trans[2];
        float sy = x*trans[3] + y*trans[4] + trans[5];
        if (sx < 0 || sy < 0 || sx >= W-1 || sy >= H-1){
            ret[tid*3+0] = Xt[tid*3+0];
            ret[tid*3+1] = Xt[tid*3+1];
            ret[tid*3+2] = Xt[tid*3+2];
            tid += offset;
            continue;
        }

        float xp = sx - (int)sx;
        float yp = sy - (int)sy;
        float color[3] = {0};
        for(int i=0;i<3;i++){
            float v = 0;
            float a = Yst[i*H*W + int(sy)*W + (int)(sx)];
            float b = Yst[i*H*W + int(sy)*W + (int)(sx+1)];
            float c = Yst[i*H*W + int(sy+1)*W + (int)(sx)];
            float d = Yst[i*H*W + int(sy+1)*W + (int)(sx+1)];
            float x1 = a + (b-a)*xp; float x2 = c + (d-c)*xp;
            v = x1 + (x2-x1)*yp;
            color[i] = v * 0.5 + 0.5;
        }
        float alpha = 0;
        {
            float a = mask[(int)(sy)*W + (int)(sx)];
            float b = mask[(int)(sy)*W + (int)(sx+1)];
            float c = mask[(int)(sy+1)*W + (int)(sx)];
            float d = mask[(int)(sy+1)*W + (int)(sx+1)];
            float x1 = a + (b-a)*xp; float x2 = c + (d-c)*xp;
            alpha = x1 + (x2-x1)*yp;
        }

        for(int i=0;i<3;i++){
            float c = color[i]*255*alpha + Xt[tid*3+(i)]*(1-alpha);
            c = c < 0 ? 0 : c;
            c = c > 255 ? 255 : c;
            ret[tid*3+i] = c;
        }

        tid += offset;
    }
}

void restore_image(uchar* ret, const float* Yst, const float* mask, const uchar* Xt,
        const float* trans, const int H, const int W, const int h, const int w) {
    restore_kernel<<<1000, 64>>>(ret, Yst, mask, Xt, trans, H, W, h, w);
    hipDeviceSynchronize();
}
